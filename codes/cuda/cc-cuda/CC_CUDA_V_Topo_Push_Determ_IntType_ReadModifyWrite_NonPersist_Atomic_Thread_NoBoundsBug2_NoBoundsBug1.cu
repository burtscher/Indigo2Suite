#include "hip/hip_runtime.h"
/*
This file is part of the Indigo2 benchmark suite version 1.0.

Copyright (c) 2023, Yiqian Liu, Noushin Azami, Avery Vanausdal, and Martin Burtscher

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice, this
   list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimer in the documentation
   and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
   contributors may be used to endorse or promote products derived from
   this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

URL: The latest version of this code is available at https://cs.txstate.edu/~burtscher/research/Indigo2Suite/.

Publication: This work is described in detail in the following paper.

Yiqian Liu, Noushin Azami, Avery Vanausdal, and Martin Burtscher. "Choosing the Best Parallelization and Implementation Styles for Graph Analytics Codes: Lessons Learned from 1106 Programs." Proceedings of the 2023 ACM/IEEE International Conference for High Performance Computing, Networking, Storage, and Analysis. November 2023.
*/

#include <cuda/atomic>

typedef int flag_t;
typedef int data_type;
typedef int basic_t;
#include "cc_vertex_cuda.h"

static const int ThreadsPerBlock = 512;

static __global__ void init(data_type* const label, data_type* const label_n, const int size)
{
  // initialize arrays
  const int v = threadIdx.x + blockIdx.x * ThreadsPerBlock;
  if (v < size) {
    label_n[v] = v;
    label[v] = v;
  }
}

static __global__ void cc(const ECLgraph g, data_type* const label, data_type* const label_n, flag_t* const goagain)
{
  int v = threadIdx.x + blockIdx.x * ThreadsPerBlock;
  if (v < g.nodes) {

    const int beg = g.nindex[v];
    const int end = g.nindex[v + 1];
    const data_type new_label = label[v];

    bool updated = false;
    for (int i = beg; i < end; i++) {
      const int dst = g.nlist[i];
      if (atomicMin(&label_n[dst], new_label) > new_label) {
        updated = true;
      }
    }
    if (updated) {
      atomicWrite(goagain, 1);
    }
  }
}

static double GPUcc_vertex(const ECLgraph& g, basic_t* const label)
{
  flag_t* d_goagain;
  data_type* d_label;
  if (hipSuccess != hipMalloc((void **)&d_goagain, sizeof(flag_t))) fprintf(stderr, "ERROR: could not allocate d_goagain\n");
  if (hipSuccess != hipMalloc((void **)&d_label, g.nodes * sizeof(data_type))) fprintf(stderr, "ERROR: could not allocate d_label\n");
  data_type* d_label_new;
  if (hipSuccess != hipMalloc((void **)&d_label_new, g.nodes * sizeof(data_type))) fprintf(stderr, "ERROR: could not allocate d_label_new\n");

  const int blocks = (g.nodes + ThreadsPerBlock - 1) / ThreadsPerBlock;

  init<<<blocks, ThreadsPerBlock>>>(d_label, d_label_new, g.nodes);

  // iterate until no more changes
  int goagain;
  int iter = 0;
  timeval start, end;
  gettimeofday(&start, NULL);

  do {
    iter++;
    goagain = 0;
    if (hipSuccess != hipMemcpy(d_goagain, &goagain, sizeof(flag_t), hipMemcpyHostToDevice)) fprintf(stderr, "ERROR: copying of go_again to device failed\n");

    cc<<<blocks, ThreadsPerBlock>>>(g, d_label, d_label_new, d_goagain);

    if (hipSuccess != hipMemcpy(&goagain, d_goagain, sizeof(flag_t), hipMemcpyDeviceToHost)) fprintf(stderr, "ERROR: copying of go_again from device failed\n");
    std::swap(d_label, d_label_new);
  } while (goagain);

  hipDeviceSynchronize();
  gettimeofday(&end, NULL);
  double runtime = end.tv_sec + end.tv_usec / 1000000.0 - start.tv_sec - start.tv_usec / 1000000.0;
  printf("iterations: %d\n", iter);

  CheckCuda();
  if (hipSuccess != hipMemcpy(label, d_label_new, g.nodes * sizeof(data_type), hipMemcpyDeviceToHost)) fprintf(stderr, "ERROR: copying of label from device failed\n");

  hipFree(d_goagain);
  hipFree(d_label);
  return runtime;
}
