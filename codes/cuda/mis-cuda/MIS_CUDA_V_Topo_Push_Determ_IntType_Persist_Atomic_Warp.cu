#include "hip/hip_runtime.h"
/*
This file is part of the Indigo2 benchmark suite version 0.9.

Copyright (c) 2023, Yiqian Liu, Noushin Azami, Avery Vanausdal, and Martin Burtscher

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice, this
   list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimer in the documentation
   and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
   contributors may be used to endorse or promote products derived from
   this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

URL: The latest version of this code is available at https://cs.txstate.edu/~burtscher/research/Indigo2Suite/.

Publication: This work is described in detail in the following paper.

Yiqian Liu, Noushin Azami, Avery Vanausdal, and Martin Burtscher. "Choosing the Best Parallelization and Implementation Styles for Graph Analytics Codes: Lessons Learned from 1106 Programs." Proceedings of the 2023 ACM/IEEE International Conference for High Performance Computing, Networking, Storage, and Analysis. November 2023.
*/

#include <cuda/atomic>
typedef int flag_t;
typedef int data_type;
static const int ThreadsPerBlock = 512;
static const int WarpSize = 32;

#include "mis_vertex_cuda.h"

static __global__ void init(data_type* const priority, flag_t* const status, flag_t* const status_n, const int size)
{
  // initialize arrays
  const int v = threadIdx.x + blockIdx.x * ThreadsPerBlock;
  if (v < size)
  {
    priority[v] = hash(v + 712313887);
    status[v] = undecided;
    status_n[v] = undecided;
  }
}

static __global__ void mis(const ECLgraph g, const data_type* const priority, flag_t* const status, flag_t* const status_n, flag_t* const goagain)
{
  const int lane = threadIdx.x % WarpSize;
  // go over all the nodes
  int tid = (threadIdx.x + blockIdx.x * ThreadsPerBlock) / WarpSize;
  for (int v = tid; v < g.nodes; v += gridDim.x * (ThreadsPerBlock / WarpSize)) {

    if (__any_sync(~0, (lane == 0) && (atomicRead(&status[v]) == undecided))) {
      int i = g.nindex[v];
      // try to find a non-excluded neighbor whose priority is higher
      if (lane == 0) {
        while ((i < g.nindex[v + 1]) && ((atomicRead(&status[g.nlist[i]]) == excluded) || (priority[v] > priority[g.nlist[i]]) || ((priority[v] == priority[g.nlist[i]]) && (v > g.nlist[i])))) {
          i++;
        }
      }
      if (__any_sync(~0, (lane == 0) && (i < g.nindex[v + 1]))) {
        // found such a neighbor -> status still unknown
        if (lane == 0) {
          atomicWrite(goagain, 1);
        }
      } else {
        // no such neighbor -> all neighbors are "excluded" and v is "included"
        if (lane == 0) {
          atomicWrite(&status_n[v], included);
        }
        for (int j = g.nindex[v] + lane; j < g.nindex[v + 1]; j += WarpSize) {
          atomicWrite(&status_n[g.nlist[j]], excluded);
        }
      }
    }
  }
}

static double GPUmis_vertex(const ECLgraph& g, data_type* const priority, int* const status)
{
  flag_t* d_goagain;
  if (hipSuccess != hipMalloc((void **)&d_goagain, sizeof(flag_t))) fprintf(stderr, "ERROR: could not allocate d_goagain\n");
  data_type* d_priority;
  if (hipSuccess != hipMalloc((void **)&d_priority, g.nodes * sizeof(data_type))) fprintf(stderr, "ERROR: could not allocate d_priority\n");
  flag_t* d_status;
  if (hipSuccess != hipMalloc((void **)&d_status, g.nodes * sizeof(flag_t))) fprintf(stderr, "ERROR: could not allocate d_status\n");
  flag_t* d_status_new;
  if (hipSuccess != hipMalloc((void **)&d_status_new, g.nodes * sizeof(flag_t))) fprintf(stderr, "ERROR: could not allocate d_status_new\n");

  const int ThreadsBound = GPUinfo(0, false);
  const int blocks = ThreadsBound / ThreadsPerBlock;

  init<<<(g.nodes + ThreadsPerBlock - 1) / ThreadsPerBlock, ThreadsPerBlock>>>(d_priority, d_status, d_status_new, g.nodes);

  timeval beg, end;
  gettimeofday(&beg, NULL);

  flag_t goagain;
  int iter = 0;
  do {
    iter++;
    goagain = 0;
    if (hipSuccess != hipMemcpy(d_goagain, &goagain, sizeof(flag_t), hipMemcpyHostToDevice)) fprintf(stderr, "ERROR: copying of goagain to device failed\n");

    mis<<<blocks, ThreadsPerBlock>>>(g, d_priority, d_status, d_status_new, d_goagain);

    if (hipSuccess != hipMemcpy(d_status, d_status_new, g.nodes * sizeof(flag_t), hipMemcpyDeviceToDevice)) fprintf(stderr, "ERROR: copying of d_status_new to d_status on device failed\n");
    if (hipSuccess != hipMemcpy(&goagain, d_goagain, sizeof(flag_t), hipMemcpyDeviceToHost)) fprintf(stderr, "ERROR: copying of goagain from device failed\n");
  } while (goagain);

  hipDeviceSynchronize();
  gettimeofday(&end, NULL);
  const double runtime = end.tv_sec - beg.tv_sec + (end.tv_usec - beg.tv_usec) / 1000000.0;

  CheckCuda();
  if (hipSuccess != hipMemcpy(status, d_status, g.nodes * sizeof(flag_t), hipMemcpyDeviceToHost)) fprintf(stderr, "ERROR: copying of status from device failed\n");

  // determine and print set size
  int cnt = 0;
  for (int v = 0; v < g.nodes; v++) {
    if (status[v] == included) cnt++;
  }
  printf("iterations: %d,  elements in set: %d (%.1f%%)\n", iter, cnt, 100.0 * cnt / g.nodes);

  hipFree(d_status_new);
  hipFree(d_goagain);
  hipFree(d_status);
  hipFree(d_priority);
  return runtime;
}
