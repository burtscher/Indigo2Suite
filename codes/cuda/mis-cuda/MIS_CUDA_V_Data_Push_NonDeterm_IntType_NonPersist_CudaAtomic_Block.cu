#include "hip/hip_runtime.h"
/*
This file is part of the Indigo2 benchmark suite version 1.0.

Copyright (c) 2023, Yiqian Liu, Noushin Azami, Avery Vanausdal, and Martin Burtscher

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice, this
   list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimer in the documentation
   and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
   contributors may be used to endorse or promote products derived from
   this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

URL: The latest version of this code is available at https://cs.txstate.edu/~burtscher/research/Indigo2Suite/.

Publication: This work is described in detail in the following paper.

Yiqian Liu, Noushin Azami, Avery Vanausdal, and Martin Burtscher. "Choosing the Best Parallelization and Implementation Styles for Graph Analytics Codes: Lessons Learned from 1106 Programs." Proceedings of the 2023 ACM/IEEE International Conference for High Performance Computing, Networking, Storage, and Analysis. November 2023.
*/

#include <cuda/atomic>
typedef cuda::atomic<int> flag_t;
typedef int data_type;
static const int ThreadsPerBlock = 512;

#include "mis_vertex_cuda.h"

static __global__ void init(data_type* const priority, flag_t* const status, const int size, int* const wl1, int* const wlsize)
{
  // initialize arrays
  const int v = threadIdx.x + blockIdx.x * ThreadsPerBlock;
  if (v < size)
  {
    priority[v] = hash(v + 712313887);
    status[v] = undecided;

    // initialize worklist
    wl1[v] = v;
  }
  if (v == 0) {
    *wlsize = size;
  }
}

static __global__ void mis(const ECLgraph g, const data_type* const priority, flag_t* const status, const int* const wl1, const int wl1size, int* const wl2, int* const wl2size)
{
  // go over all nodes in worklist
  int w = blockIdx.x;
  if (w < wl1size) {

    int v = wl1[w];
    if (__syncthreads_or((threadIdx.x == 0) && (status[v].load() == undecided))) {
      int i = g.nindex[v];
      // try to find a non-excluded neighbor whose priority is higher
      if (threadIdx.x == 0) {
        while ((i < g.nindex[v + 1]) && ((status[g.nlist[i]].load() == excluded) || (priority[v] > priority[g.nlist[i]]) || ((priority[v] == priority[g.nlist[i]]) && (v > g.nlist[i])))) {
          i++;
        }
      }
      if (__syncthreads_or((threadIdx.x == 0) && (i < g.nindex[v + 1]))) {
        // found such a neighbor -> status still unknown
        if (threadIdx.x == 0) {
          wl2[atomicAdd(wl2size, 1)] = v;
        }
      } else {
        // no such neighbor -> all neighbors are "excluded" and v is "included"
        if (threadIdx.x == 0) {
          status[v].store(included);
        }
        for (int j = g.nindex[v] + threadIdx.x; j < g.nindex[v + 1]; j += ThreadsPerBlock) {
          status[g.nlist[j]].store(excluded);
        }
      }
    }
  }
}

static double GPUmis_vertex(const ECLgraph& g, data_type* const priority, int* const status)
{
  data_type* d_priority;
  if (hipSuccess != hipMalloc((void **)&d_priority, g.nodes * sizeof(data_type))) fprintf(stderr, "ERROR: could not allocate d_priority\n");
  flag_t* d_status;
  if (hipSuccess != hipMalloc((void **)&d_status, g.nodes * sizeof(flag_t))) fprintf(stderr, "ERROR: could not allocate d_status\n");

  int* d_wl1;
  if (hipSuccess != hipMalloc((void **)&d_wl1, g.nodes * sizeof(int))) fprintf(stderr, "ERROR: could not allocate d_wl1\n");
  int* d_wl1size;
  if (hipSuccess != hipMalloc((void **)&d_wl1size, sizeof(int))) fprintf(stderr, "ERROR: could not allocate d_wl1size\n");
  int* d_wl2;
  if (hipSuccess != hipMalloc((void **)&d_wl2, g.nodes * sizeof(int))) fprintf(stderr, "ERROR: could not allocate d_wl2\n");
  int* d_wl2size;
  if (hipSuccess != hipMalloc((void **)&d_wl2size, sizeof(int))) fprintf(stderr, "ERROR: could not allocate d_wl2size\n");
  int wlsize;


  init<<<(g.nodes + ThreadsPerBlock - 1) / ThreadsPerBlock, ThreadsPerBlock>>>(d_priority, d_status, g.nodes, d_wl1, d_wl1size);

  if (hipSuccess != hipMemcpy(&wlsize, d_wl1size, sizeof(int), hipMemcpyDeviceToHost)) fprintf(stderr, "ERROR: copying of wlsize from device d_wl1size failed\n");

  timeval beg, end;
  gettimeofday(&beg, NULL);

  int iter = 0;
  do {
    iter++;
    hipMemset(d_wl2size, 0, sizeof(int));
    const int blocks = wlsize;

    mis<<<blocks, ThreadsPerBlock>>>(g, d_priority, d_status, d_wl1, wlsize, d_wl2, d_wl2size);

    if (hipSuccess != hipMemcpy(&wlsize, d_wl2size, sizeof(int), hipMemcpyDeviceToHost)) { fprintf(stderr, "ERROR: copying of wlsize from device failed\n"); break; }
    std::swap(d_wl1, d_wl2);
    std::swap(d_wl1size, d_wl2size);
  } while (wlsize > 0);

  hipDeviceSynchronize();
  gettimeofday(&end, NULL);
  const double runtime = end.tv_sec - beg.tv_sec + (end.tv_usec - beg.tv_usec) / 1000000.0;

  CheckCuda();
  if (hipSuccess != hipMemcpy(status, d_status, g.nodes * sizeof(flag_t), hipMemcpyDeviceToHost)) fprintf(stderr, "ERROR: copying of status from device failed\n");

  // determine and print set size
  int cnt = 0;
  for (int v = 0; v < g.nodes; v++) {
    if (status[v] == included) cnt++;
  }
  printf("iterations: %d,  elements in set: %d (%.1f%%)\n", iter, cnt, 100.0 * cnt / g.nodes);

  hipFree(d_status);
  hipFree(d_priority);
  hipFree(d_wl1);
  hipFree(d_wl1size);
  hipFree(d_wl2);
  hipFree(d_wl2size);
  return runtime;
}
