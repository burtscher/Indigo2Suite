#include "hip/hip_runtime.h"
/*
This file is part of the Indigo2 benchmark suite version 1.0.

Copyright (c) 2023, Yiqian Liu, Noushin Azami, Avery Vanausdal, and Martin Burtscher

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice, this
   list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimer in the documentation
   and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
   contributors may be used to endorse or promote products derived from
   this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

URL: The latest version of this code is available at https://cs.txstate.edu/~burtscher/research/Indigo2Suite/.

Publication: This work is described in detail in the following paper.

Yiqian Liu, Noushin Azami, Avery Vanausdal, and Martin Burtscher. "Choosing the Best Parallelization and Implementation Styles for Graph Analytics Codes: Lessons Learned from 1106 Programs." Proceedings of the 2023 ACM/IEEE International Conference for High Performance Computing, Networking, Storage, and Analysis. November 2023.
*/

#include <cuda/atomic>
typedef cuda::atomic<int> flag_t;
typedef int data_type;
static const int ThreadsPerBlock = 512;

#include "mis_edge_cuda.h"

static __global__ void init(const ECLgraph g, const int* const sp, data_type* const priority, flag_t* const status, flag_t* const status_n, flag_t* const lost, int* const wl1, int* const wlsize)
{
  // initialize arrays
  const int v = threadIdx.x + blockIdx.x * ThreadsPerBlock;
  if (v < g.nodes)
  {
    priority[v] = hash(v + 712313887);
    status[v] = undecided;
    status_n[v] = undecided;
    lost[v] = 0;
  }
  if (v < g.edges)
  {
    // initialize worklist
    if (sp[v] < g.nlist[v]) {
      wl1[atomicAdd(wlsize, 1)] = v;
    }
  }
}

static __global__ void mis(const ECLgraph g, const int* const sp, const data_type* const priority, flag_t* const status, flag_t* const status_n, flag_t* const lost, const int* const wl1, const int wl1size)
{
  // go over all edges in wl1
  int w = threadIdx.x + blockIdx.x * ThreadsPerBlock;
  if (w < wl1size) {

    int e = wl1[w];
    const int src = sp[e];
    const int dst = g.nlist[e];
    const int srcStatus = status[src].load();
    const int dstStatus = status[dst].load();

    // if one is included, exclude the other
    if (srcStatus == included) {
      status_n[dst].store(excluded);
    }
    else if (dstStatus == included) {
      status_n[src].store(excluded);
    } else if (srcStatus == undecided && dstStatus == undecided) {
      // if both undecided -> mark lower as lost
      if (priority[src] < priority[dst]) {
        lost[src].store(1);
      } else {
        lost[dst].store(1);
      }
    }
  }
}

static __global__ void mis_vertex_pass(const ECLgraph g, const int* const sp, data_type* const priority, flag_t* const status, flag_t* const status_n, flag_t* const lost, const int* const wl1, const int wl1size, int* const wl2, int* const wl2size, const int iter, int* const time)
{
  // go over all edges in wl1 and check if lost
  int w = threadIdx.x + blockIdx.x * ThreadsPerBlock;
  if (w < wl1size) {

    const int e = wl1[w];
    const int src = sp[e];
    const int dst = g.nlist[e];
    const int srcStatus = status[src].load();
    const int dstStatus = status[dst].load();

    // if src won
    if (lost[src] == 0) {
      if (srcStatus == undecided) {
        // and is undecided -> include
        status_n[src].store(included);
      }
    }
    // if dst won
    if (lost[dst] == 0) {
      if (dstStatus == undecided) {
        // and is undecided -> include
        status_n[dst].store(included);
      }
    }
    // if either is still undecided, keep it in WL
    if (srcStatus == undecided || dstStatus == undecided) {
      if (atomicMax(&time[e], iter) < iter) {
        wl2[atomicAdd(wl2size, 1)] = e;
      }
    }
  }
}

static __global__ void mis_last_pass(flag_t* const status, const int size)
{
  int w = threadIdx.x + blockIdx.x * ThreadsPerBlock;
  if (w < size) {
    if (status[w] == undecided)
    {
      status[w] = included;
    }
  }
}

static double GPUmis_edge(const ECLgraph& g, const int* const sp, data_type* const priority, int* const status)
{
  data_type* d_priority;
  if (hipSuccess != hipMalloc((void **)&d_priority, g.nodes * sizeof(data_type))) fprintf(stderr, "ERROR: could not allocate d_priority\n");
  flag_t* d_status;
  if (hipSuccess != hipMalloc((void **)&d_status, g.nodes * sizeof(flag_t))) fprintf(stderr, "ERROR: could not allocate d_status\n");
  flag_t* d_lost;
  if (hipSuccess != hipMalloc((void **)&d_lost, g.nodes * sizeof(flag_t))) fprintf(stderr, "ERROR: could not allocate d_lost\n");
  flag_t* d_status_new;
  if (hipSuccess != hipMalloc((void **)&d_status_new, g.nodes * sizeof(flag_t))) fprintf(stderr, "ERROR: could not allocate d_status_new\n");

  int* d_wl1;
  if (hipSuccess != hipMalloc((void **)&d_wl1, std::max(g.edges, g.nodes) * sizeof(int))) fprintf(stderr, "ERROR: could not allocate d_wl1\n");
  int* d_wl1size;
  if (hipSuccess != hipMalloc((void **)&d_wl1size, sizeof(int))) fprintf(stderr, "ERROR: could not allocate d_wl1size\n");
  int* d_wl2;
  if (hipSuccess != hipMalloc((void **)&d_wl2, std::max(g.edges, g.nodes) * sizeof(int))) fprintf(stderr, "ERROR: could not allocate d_wl2\n");
  int* d_wl2size;
  if (hipSuccess != hipMalloc((void **)&d_wl2size, sizeof(int))) fprintf(stderr, "ERROR: could not allocate d_wl2size\n");
  int* d_time;
  if (hipSuccess != hipMalloc((void **)&d_time, sizeof(int) * g.edges)) {fprintf(stderr, "ERROR: could not allocate memory\n"); exit(-1);}
  hipMemset(d_time, 0, sizeof(int) * g.edges);
  int wlsize;
  hipMemset(d_wl1size, 0, sizeof(int));

  init<<<(g.edges + ThreadsPerBlock - 1) / ThreadsPerBlock, ThreadsPerBlock>>>(g, sp, d_priority, d_status, d_status_new, d_lost, d_wl1, d_wl1size);

  if (hipSuccess != hipMemcpy(&wlsize, d_wl1size, sizeof(int), hipMemcpyDeviceToHost)) fprintf(stderr, "ERROR: copying of wlsize from device d_wl1size failed\n");

  timeval beg, end;
  gettimeofday(&beg, NULL);

  int iter = 0;
  do {
    iter++;
    hipMemset(d_wl2size, 0, sizeof(int));
    const int blocks = (wlsize + ThreadsPerBlock - 1) / ThreadsPerBlock;

    // edge pass
    mis<<<blocks, ThreadsPerBlock>>>(g, sp, d_priority, d_status, d_status_new, d_lost, d_wl1, wlsize);

    if (hipSuccess != hipMemcpy(d_status, d_status_new, g.nodes * sizeof(flag_t), hipMemcpyDeviceToDevice)) fprintf(stderr, "ERROR: copying of d_status_new to d_status on device failed\n");
    // vertex pass
    mis_vertex_pass<<<blocks, ThreadsPerBlock>>>(g, sp, d_priority, d_status, d_status_new, d_lost, d_wl1, wlsize, d_wl2, d_wl2size, iter, d_time);

    if (hipSuccess != hipMemcpy(&wlsize, d_wl2size, sizeof(int), hipMemcpyDeviceToHost)) { fprintf(stderr, "ERROR: copying of wlsize from device failed\n"); break; }
    hipMemset(d_lost, 0, g.nodes * sizeof(flag_t));
    std::swap(d_wl1, d_wl2);
    std::swap(d_wl1size, d_wl2size);
    if (hipSuccess != hipMemcpy(d_status, d_status_new, g.nodes * sizeof(flag_t), hipMemcpyDeviceToDevice)) fprintf(stderr, "ERROR: copying of d_status_new to d_status on device failed\n");
  } while (wlsize > 0);

  const int blocks = (g.nodes + ThreadsPerBlock - 1) / ThreadsPerBlock;
  // include all remaining nodes that have no edges
  mis_last_pass<<<blocks, ThreadsPerBlock>>>(d_status, g.nodes);

  hipDeviceSynchronize();
  gettimeofday(&end, NULL);
  const double runtime = end.tv_sec - beg.tv_sec + (end.tv_usec - beg.tv_usec) / 1000000.0;

  CheckCuda();
  if (hipSuccess != hipMemcpy(status, d_status, g.nodes * sizeof(flag_t), hipMemcpyDeviceToHost)) fprintf(stderr, "ERROR: copying of status from device failed\n");

  // determine and print set size
  int cnt = 0;
  for (int v = 0; v < g.nodes; v++) {
    if (status[v] == included) cnt++;
  }
  printf("iterations: %d,  elements in set: %d (%.1f%%)\n", iter, cnt, 100.0 * cnt / g.nodes);

  hipFree(d_status_new);
  hipFree(d_status);
  hipFree(d_priority);
  hipFree(d_lost);
  hipFree(d_wl1);
  hipFree(d_wl1size);
  hipFree(d_wl2);
  hipFree(d_wl2size);
  return runtime;
}
