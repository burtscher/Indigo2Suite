#include "hip/hip_runtime.h"
/*
This file is part of the Indigo2 benchmark suite version 0.9.

Copyright (c) 2023, Yiqian Liu, Noushin Azami, Avery Vanausdal, and Martin Burtscher

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice, this
   list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimer in the documentation
   and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
   contributors may be used to endorse or promote products derived from
   this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

URL: The latest version of this code is available at https://cs.txstate.edu/~burtscher/research/Indigo2Suite/.

Publication: This work is described in detail in the following paper.

Yiqian Liu, Noushin Azami, Avery Vanausdal, and Martin Burtscher. "Choosing the Best Parallelization and Implementation Styles for Graph Analytics Codes: Lessons Learned from 1106 Programs." Proceedings of the 2023 ACM/IEEE International Conference for High Performance Computing, Networking, Storage, and Analysis. November 2023.
*/

#include <cuda/atomic>
typedef cuda::atomic<int> flag_t;
typedef cuda::atomic<int> data_type;
typedef int basic_t;
static const int ThreadsPerBlock = 512;
static const int WarpSize = 32;

#include "sssp_vertex_cuda.h"

static __global__ void init(const int src, data_type* const dist, const int size)
{
  // initialize arrays
  const int v = threadIdx.x + blockIdx.x * ThreadsPerBlock;
  if (v < size) {
    const data_type temp = (v == src) ? 0 : maxval;
    dist[v].store(temp);
  }
}

static __global__ void sssp(const ECLgraph g, data_type* const dist, flag_t* const goagain)
{
  int v = (threadIdx.x + blockIdx.x * ThreadsPerBlock) / WarpSize;
  if (v < g.nodes) {

    const int beg = g.nindex[v];
    const int end = g.nindex[v + 1];
    const data_type s = dist[v].load();

    if (s != maxval) {
      bool updated = false;
      for (int i = beg + threadIdx.x % WarpSize; i < end; i += WarpSize) {
        const int dst = g.nlist[i];
        const data_type new_dist = s + g.eweight[i];
        const data_type d = dist[dst].load();
        if (d > new_dist) {
          dist[dst].store(new_dist);
          updated = true;
        }
      }
      if (updated) {
        *goagain = 1;
      }
    }
  }
}

static double GPUsssp_vertex(const int src, const ECLgraph& g, basic_t* const dist)
{
  flag_t* d_goagain;
  data_type* d_dist;
  if (hipSuccess != hipMalloc((void **)&d_goagain, sizeof(flag_t))) fprintf(stderr, "ERROR: could not allocate d_goagain\n");
  if (hipSuccess != hipMalloc((void **)&d_dist, g.nodes * sizeof(data_type))) fprintf(stderr, "ERROR: could not allocate d_dist\n");

  const int blocks = ((long)g.nodes * WarpSize + ThreadsPerBlock - 1) / ThreadsPerBlock;

  init<<<(g.nodes + ThreadsPerBlock - 1) / ThreadsPerBlock, ThreadsPerBlock>>>(src, d_dist, g.nodes);

  // iterate until no more changes
  int goagain;
  int iter = 0;

  timeval start, end;
  gettimeofday(&start, NULL);

  do {
    iter++;
    goagain = 0;
    if (hipSuccess != hipMemcpy(d_goagain, &goagain, sizeof(flag_t), hipMemcpyHostToDevice)) fprintf(stderr, "ERROR: copying of go_again to device failed\n");

    sssp<<<blocks, ThreadsPerBlock>>>(g, d_dist, d_goagain);

    if (hipSuccess != hipMemcpy(&goagain, d_goagain, sizeof(flag_t), hipMemcpyDeviceToHost)) fprintf(stderr, "ERROR: copying of go_again from device failed\n");
  } while (goagain);

  hipDeviceSynchronize();
  gettimeofday(&end, NULL);
  double runtime = end.tv_sec + end.tv_usec / 1000000.0 - start.tv_sec - start.tv_usec / 1000000.0;
  printf("iterations: %d\n", iter);

  CheckCuda();
  if (hipSuccess != hipMemcpy(dist, d_dist, g.nodes * sizeof(data_type), hipMemcpyDeviceToHost)) fprintf(stderr, "ERROR: copying of dist from device failed\n");

  hipFree(d_goagain);
  hipFree(d_dist);
  return runtime;
}
