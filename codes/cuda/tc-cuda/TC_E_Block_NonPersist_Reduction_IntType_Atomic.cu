#include "hip/hip_runtime.h"
/*
This file is part of the Indigo2 benchmark suite version 0.9.

Copyright (c) 2023, Yiqian Liu, Noushin Azami, Avery Vanausdal, and Martin Burtscher

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice, this
   list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimer in the documentation
   and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
   contributors may be used to endorse or promote products derived from
   this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

URL: The latest version of this code is available at https://cs.txstate.edu/~burtscher/research/Indigo2Suite/.

Publication: This work is described in detail in the following paper.

Yiqian Liu, Noushin Azami, Avery Vanausdal, and Martin Burtscher. "Choosing the Best Parallelization and Implementation Styles for Graph Analytics Codes: Lessons Learned from 1106 Programs." Proceedings of the 2023 ACM/IEEE International Conference for High Performance Computing, Networking, Storage, and Analysis. November 2023.
*/

typedef int data_type;
typedef int basic_t;
static const int WS = 32;
static const int ThreadsPerBlock = 512;
#include "tc_edge_cuda.h"
static __global__ void d_triCounting(data_type* g_count, const int edges, const int* const __restrict__ nindex, const int* const __restrict__ nlist, const int* const sp)
{
  __shared__ int s_buffer[WS];
  const int lane = threadIdx.x % WS;
  const int warp = threadIdx.x / WS;
  basic_t count = 0;
  const int e = blockIdx.x;
  if (e < edges) {
    const int src = sp[e];
    const int dst = nlist[e];
    if (src > dst) {
      const int beg1 = nindex[dst];
      const int end1 = nindex[dst + 1];
      for (int i = beg1 + threadIdx.x; i < end1 && nlist[i] < dst; i += ThreadsPerBlock){
        const int u = nlist[i];
        int beg2 = nindex[src];
        int end2 = nindex[src + 1];
        if (d_find(u, beg2, end2, nlist)) count++;
      }
    }
  }
  // warp reduction
  count += __shfl_down_sync(~0, count, 16);
  count += __shfl_down_sync(~0, count, 8);
  count += __shfl_down_sync(~0, count, 4);
  count += __shfl_down_sync(~0, count, 2);
  count += __shfl_down_sync(~0, count, 1);
  if (lane == 0) s_buffer[warp] = count;
  __syncthreads();
  // block reduction
  if (warp == 0) {
    int val = s_buffer[lane];
    val += __shfl_down_sync(~0, val, 16);
    val += __shfl_down_sync(~0, val, 8);
    val += __shfl_down_sync(~0, val, 4);
    val += __shfl_down_sync(~0, val, 2);
    val += __shfl_down_sync(~0, val, 1);
    if (lane == 0) atomicAdd(g_count, val);
  }
}
static double GPUtc_edge(basic_t &count, const int edges, const int* const nindex, const int* const nlist, const int* const sp)
{
  data_type* d_count;
  if (hipSuccess != hipMalloc((void **)&d_count, sizeof(data_type))) fprintf(stderr, "ERROR: could not allocate d_goagain\n");
  timeval start, end;
  const int blocks = edges;
  count = 0;
  gettimeofday(&start, NULL);
  if (hipSuccess != hipMemcpy(d_count, &count, sizeof(data_type), hipMemcpyHostToDevice)) fprintf(stderr, "ERROR: copying of go_again to device failed\n");
  d_triCounting<<<blocks, ThreadsPerBlock>>>(d_count, edges, nindex, nlist, sp);
  if (hipSuccess != hipMemcpy(&count, d_count, sizeof(data_type), hipMemcpyDeviceToHost)) fprintf(stderr, "ERROR: copying of go_again from device failed\n");
  hipDeviceSynchronize();
  gettimeofday(&end, NULL);
  hipFree(d_count);
  return (end.tv_sec + end.tv_usec / 1000000.0 - start.tv_sec - start.tv_usec / 1000000.0);
}
