#include "hip/hip_runtime.h"
/*
This file is part of the Indigo2 benchmark suite version 0.9.

Copyright (c) 2023, Yiqian Liu, Noushin Azami, Avery Vanausdal, and Martin Burtscher

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice, this
   list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimer in the documentation
   and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
   contributors may be used to endorse or promote products derived from
   this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

URL: The latest version of this code is available at https://cs.txstate.edu/~burtscher/research/Indigo2Suite/.

Publication: This work is described in detail in the following paper.

Yiqian Liu, Noushin Azami, Avery Vanausdal, and Martin Burtscher. "Choosing the Best Parallelization and Implementation Styles for Graph Analytics Codes: Lessons Learned from 1106 Programs." Proceedings of the 2023 ACM/IEEE International Conference for High Performance Computing, Networking, Storage, and Analysis. November 2023.
*/

#include <cuda/atomic>
typedef cuda::atomic<int> flag_t;
typedef cuda::atomic<int> data_type;
typedef int basic_t;
static const int ThreadsPerBlock = 512;

#include "bfs_edge_cuda.h"

static __global__ void init(const int src, data_type* const dist, const int size)
{
  // initialize arrays
  const int v = threadIdx.x + blockIdx.x * ThreadsPerBlock;
  if (v < size) {
    const data_type temp = (v == src) ? 0 : maxval;
    dist[v].store(temp);
  }
}

static __global__ void bfs(const ECLgraph g, const int* const sp, data_type* const dist, flag_t* const goagain)
{
  int e = threadIdx.x + blockIdx.x * ThreadsPerBlock;
  if (e < g.edges) {

    const int src = sp[e];
    const int dst = g.nlist[e];
    const data_type s = dist[src].load();

    if (s != maxval) {
      const data_type new_dist = s + 1;
      data_type d = dist[dst].load();
      if (d > new_dist) {
        dist[dst].store(new_dist);
        *goagain = 1;
      }
    }
  }
}

static double GPUbfs_edge(const int src, const ECLgraph& g, basic_t* const dist, const int* const sp)
{
  flag_t* d_goagain;
  data_type* d_dist;
  int* d_sp;
  if (hipSuccess != hipMalloc((void **)&d_goagain, sizeof(flag_t))) fprintf(stderr, "ERROR: could not allocate d_goagain\n");
  if (hipSuccess != hipMalloc((void **)&d_dist, g.nodes * sizeof(data_type))) fprintf(stderr, "ERROR: could not allocate d_dist\n");
  if (hipSuccess != hipMalloc((void **)&d_sp, sizeof(int) * g.edges)) {fprintf(stderr, "ERROR: could not allocate d_sp\n"); exit(-1);}
  hipMemcpy(d_sp, sp, sizeof(int) * g.edges, hipMemcpyHostToDevice);

  const int blocks = (g.edges + ThreadsPerBlock - 1) / ThreadsPerBlock;

  timeval start, end;
  init<<<(g.nodes + ThreadsPerBlock - 1) / ThreadsPerBlock, ThreadsPerBlock>>>(src, d_dist, g.nodes);

  // iterate until no more changes
  int goagain;
  int iter = 0;
  gettimeofday(&start, NULL);

  do {
    iter++;
    goagain = 0;
    if (hipSuccess != hipMemcpy(d_goagain, &goagain, sizeof(int), hipMemcpyHostToDevice)) fprintf(stderr, "ERROR: copying of go_again to device failed\n");

    bfs<<<blocks, ThreadsPerBlock>>>(g, d_sp, d_dist, d_goagain);

    if (hipSuccess != hipMemcpy(&goagain, d_goagain, sizeof(int), hipMemcpyDeviceToHost)) fprintf(stderr, "ERROR: copying of go_again from device failed\n");
  } while (goagain);

  hipDeviceSynchronize();
  gettimeofday(&end, NULL);
  double runtime = end.tv_sec + end.tv_usec / 1000000.0 - start.tv_sec - start.tv_usec / 1000000.0;
  printf("iterations: %d\n", iter);

  CheckCuda();
  if (hipSuccess != hipMemcpy(dist, d_dist, g.nodes * sizeof(data_type), hipMemcpyDeviceToHost)) fprintf(stderr, "ERROR: copying of dist from device failed\n");

  hipFree(d_goagain);
  hipFree(d_dist);
  return runtime;
}
