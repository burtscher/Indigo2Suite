#include "hip/hip_runtime.h"
/*
This file is part of the Indigo2 benchmark suite version 0.9.

Copyright (c) 2023, Yiqian Liu, Noushin Azami, Avery Vanausdal, and Martin Burtscher

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice, this
   list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimer in the documentation
   and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
   contributors may be used to endorse or promote products derived from
   this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

URL: The latest version of this code is available at https://cs.txstate.edu/~burtscher/research/Indigo2Suite/.

Publication: This work is described in detail in the following paper.

Yiqian Liu, Noushin Azami, Avery Vanausdal, and Martin Burtscher. "Choosing the Best Parallelization and Implementation Styles for Graph Analytics Codes: Lessons Learned from 1106 Programs." Proceedings of the 2023 ACM/IEEE International Conference for High Performance Computing, Networking, Storage, and Analysis. November 2023.
*/

#include <cuda/atomic>
typedef cuda::atomic<int> flag_t;
typedef cuda::atomic<int> data_type;
typedef int basic_t;
static const int ThreadsPerBlock = 512;

#include "bfs_vertex_cuda.h"

static __global__ void init(const int src, data_type* const dist, data_type* const dist_n, const int size, const ECLgraph g, int* const wl1, int* const wlsize)
{
  // initialize dist array
  const int v = threadIdx.x + blockIdx.x * ThreadsPerBlock;
  if (v < size) {
    const data_type temp = (v == src) ? 0 : maxval;
    dist_n[v].store(temp);
    dist[v].store(temp);
  }
  // initialize worklist
  if (v == 0) {
    wl1[0] = src;
    *wlsize = 1;
  }
}

static __global__ void bfs_vertex_data(const ECLgraph g, data_type* const dist, data_type* const dist_n, const int* const wl1, const int wl1size, int* const wl2, int* const wl2size, const int iter, int* const time)
{
  int tid = threadIdx.x + blockIdx.x * ThreadsPerBlock;
  for (int idx = tid; idx < wl1size; idx += gridDim.x * ThreadsPerBlock) {
    const int src = wl1[idx];
    const data_type s = dist[src].load();
    const int beg = g.nindex[src];
    const int end = g.nindex[src + 1];

    if (s != maxval) {
      for (int i = beg; i < end; i++) {
        const int dst = g.nlist[i];
        const data_type new_dist = s + 1;

        if (dist_n[dst].fetch_min(new_dist) > new_dist) {
          if (atomicMax(&time[dst], iter) != iter) {
            wl2[atomicAdd(wl2size, 1)] = dst;
          }
        }
      }
      dist_n[src].fetch_min(s, cuda::memory_order_relaxed);
    }
  }
}
static double GPUbfs_vertex(const int src, const ECLgraph& g, basic_t* const dist)
{
  data_type* d_dist;
  if (hipSuccess != hipMalloc((void **)&d_dist, g.nodes * sizeof(data_type))) fprintf(stderr, "ERROR: could not allocate d_dist\n");
  data_type* d_dist_new;
  if (hipSuccess != hipMalloc((void **)&d_dist_new, g.nodes * sizeof(data_type))) fprintf(stderr, "ERROR: could not allocate d_dist_new\n");
  int* d_wl1;
  if (hipSuccess != hipMalloc((void **)&d_wl1, std::max(g.edges, g.nodes) * sizeof(int))) fprintf(stderr, "ERROR: could not allocate d_wl1\n");
  int* d_wl1size;
  if (hipSuccess != hipMalloc((void **)&d_wl1size, sizeof(int))) fprintf(stderr, "ERROR: could not allocate d_wl1size\n");
  int* d_wl2;
  if (hipSuccess != hipMalloc((void **)&d_wl2, std::max(g.edges, g.nodes) * sizeof(int))) fprintf(stderr, "ERROR: could not allocate d_wl2\n");
  int* d_wl2size;
  if (hipSuccess != hipMalloc((void **)&d_wl2size, sizeof(int))) fprintf(stderr, "ERROR: could not allocate d_wl2size\n");
  int* d_time;
  if (hipSuccess != hipMalloc((void **)&d_time, sizeof(int) * std::max(g.edges, g.nodes))) {fprintf(stderr, "ERROR: could not allocate memory\n"); exit(-1);}
  hipMemset(d_time, 0, sizeof(int) * g.nodes);
  int wlsize;
  const int ThreadsBound = GPUinfo(0);
  const int blocks = ThreadsBound / ThreadsPerBlock;

  timeval start, end;

  init<<<(g.nodes + ThreadsPerBlock - 1) / ThreadsPerBlock, ThreadsPerBlock>>>(src, d_dist, d_dist_new, g.nodes, g, d_wl1, d_wl2size);

  if (hipSuccess != hipMemcpy(&wlsize, d_wl2size, sizeof(int), hipMemcpyDeviceToHost)) fprintf(stderr, "ERROR: copying of wlsize from device failed\n");
  if (hipSuccess != hipMemcpy(d_wl1size, &wlsize, sizeof(int), hipMemcpyHostToDevice)) fprintf(stderr, "ERROR: copying of wl1size to device failed\n");
  // iterate until no more changes
  int iter = 0;
  gettimeofday(&start, NULL);

  do {
    iter++;
    hipMemset(d_wl2size, 0, sizeof(int));

    bfs_vertex_data<<<blocks, ThreadsPerBlock>>>(g, d_dist, d_dist_new, d_wl1, wlsize, d_wl2, d_wl2size, iter, d_time);

    if (hipSuccess != hipMemcpy(&wlsize, d_wl2size, sizeof(int), hipMemcpyDeviceToHost)) fprintf(stderr, "ERROR: copying of wlsize from device failed\n");
    std::swap(d_wl1, d_wl2);
    std::swap(d_wl1size, d_wl2size);
    std::swap(d_dist, d_dist_new);
  } while (wlsize > 0);

  hipDeviceSynchronize();
  gettimeofday(&end, NULL);

  double runtime = end.tv_sec + end.tv_usec / 1000000.0 - start.tv_sec - start.tv_usec / 1000000.0;
  CheckCuda();
  printf("iterations: %d\n", iter);

  if (hipSuccess != hipMemcpy(dist, d_dist_new, g.nodes * sizeof(data_type), hipMemcpyDeviceToHost)) fprintf(stderr, "ERROR: copying of dist from device failed\n");

  hipFree(d_dist);
  hipFree(d_wl1);
  hipFree(d_wl1size);
  hipFree(d_wl2);
  hipFree(d_wl2size);
  hipFree(d_time);
  return runtime;
}
